
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <stdlib.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>

constexpr uint32_t THREADS_PER_BLOCK = 512;
